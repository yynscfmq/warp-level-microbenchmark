#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define DATATYPE int
#define MEMSIZE 1024
#define REP 128
#define conflictnum 32

__constant__ int d_array_m1[MEMSIZE];
__constant__ int d_array_m2[MEMSIZE];
__global__ void constant_broadcast(double *time,DATATYPE *out,int its)
{
	DATATYPE p,q=(threadIdx.x/conflictnum);
//	DATATYPE p,q=(threadIdx.x/conflictnum*conflictnum);
	double time_tmp=0.0;
	unsigned int start_time=0,stop_time=0;
	unsigned int i,j;
	for (i=0;i<its;i++)
	{
		__syncthreads();
		start_time=clock();
#pragma unroll
		for (j=0;j<REP;j++)
		{
			p=d_array_m1[q];
			q=d_array_m2[p];
		}
		stop_time=clock();
		time_tmp+=(stop_time-start_time);
	}
	time_tmp=time_tmp/REP/its;
	out[blockDim.x*blockIdx.x+threadIdx.x] = p+q;
	time[blockDim.x*blockIdx.x+threadIdx.x] = time_tmp;
}

int main_test(int blocks,int threads,DATATYPE *h_in1,DATATYPE *h_in2)
{
	int its=30;
	//int blocks=1,threads=32;
	hipMemcpyToSymbol(HIP_SYMBOL(d_array_m1),h_in1,MEMSIZE*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_array_m2),h_in2,MEMSIZE*sizeof(int),0,hipMemcpyHostToDevice);
	double *h_time,*d_time;
	DATATYPE *d_out;
	h_time=(double*)malloc(sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_time,sizeof(double)*blocks*threads);
	hipMalloc((void**)&d_out,sizeof(DATATYPE)*blocks*threads);

	constant_broadcast<<<blocks,threads>>>(d_time,d_out,its);
	hipMemcpy(h_time,d_time,sizeof(double)*blocks*threads,hipMemcpyDeviceToHost);
	double avert=0.0,maxt=0.0,mint=99999.9;
	int nn=0;
	for (int i=0;i<blocks;i++)
	{
		for (int j=0;j<threads;j+=32)
		{
			avert+=h_time[i*threads+j];
			nn++;
			if (maxt<h_time[i*threads+j])
			{
				maxt=h_time[i*threads+j];
			}
			if (mint>h_time[i*threads+j])
			{
				mint=h_time[i*threads+j];
			}
		}
	}
	avert/=nn;
	printf("%d\t%d\t\t%f\t%f\t%f\n", blocks,threads,avert,mint,maxt);
	hipFree(d_time);
	hipFree(d_out);
	free(h_time);
	return 0;
}
void init_order(DATATYPE *a,int n)
{
	for (int i=0;i<n;i++)
	{
		a[i]=i;
	}
}

int main()
{
	DATATYPE *h_in1;
	h_in1=(DATATYPE*)malloc(sizeof(DATATYPE)*MEMSIZE);

	init_order(h_in1,MEMSIZE);


/*
	for (int i=0;i<MEMSIZE;i+=32)
	{
		for (int j=0;j<32;j++)
		{
			printf("%d\t",h_in3[i+j]);
		}
		printf("\n");
	}
*/

	printf("blocks\t threads\t aver \t min \t max \t(clocks)\n");

	//main_test(1,32,h_in1,h_in1,1);
	//main_test(1,32,h_in2,h_in2,2);
	//main_test(1,32,h_in3,h_in3,3);
	//main_test(1,512,h_in1,h_in1,1);
	//main_test(1,512,h_in2,h_in2,2);
	//main_test(1,512,h_in3,h_in3,3);



/*
	for (int i=0;i<=1;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=0;j<=512;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}
*/





	for (int i=0;i<=1024;i+=32)
	{
		int blocks=i;
		if (i==0)
		{
			blocks=1;
		}
		for (int j=1024;j<=1024;j+=32)
		{
			int threads=j;
			if (j==0)
			{
				threads=1;
			}
			main_test(blocks,threads,h_in1,h_in1);
		}
	}



	free(h_in1);

	return 0;
}
